#include "hip/hip_runtime.h"
#include <torch/torch.h>

#include "cuda_error.hpp"
#include "error.hpp"

namespace tenviz {
template <typename scalar_t>
__global__ void KernIndexPut(
    const torch::PackedTensorAccessor<int64_t, 1, torch::RestrictPtrTraits,
                                      size_t>
        indices,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits,
                                      size_t>
        tdata,
    scalar_t *bdata) {
  const int indices_idx = blockIdx.x;
  if (indices_idx < indices.size(0)) {
    const int idx = indices[indices_idx];
    const int col_idx = threadIdx.x;
    if (col_idx < tdata.size(1)) {
      const size_t offset = tdata.size(1) * idx + col_idx;
      bdata[offset] = tdata[indices_idx][col_idx];
    }
  }
}

template <typename scalar_t>
__global__ void KernIndexSelect(
    const torch::PackedTensorAccessor<int64_t, 1, torch::RestrictPtrTraits,
                                      size_t>
        indices,
    const scalar_t *gl_data,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>
        tensor) {
  const int tensor_idx = blockIdx.x;
  if (tensor_idx < indices.size(0)) {
    const int col_idx = threadIdx.x;
    if (col_idx < tensor.size(1)) {
      const int buffer_idx = indices[tensor_idx];
      const int offset = tensor.size(1) * buffer_idx + col_idx;
      tensor[tensor_idx][col_idx] = gl_data[offset];
    }
  }
}

void CUDAIndexPut(const torch::Tensor &indices, const torch::Tensor &tensor,
                  void *buffer_data) {
  int blk_sz = indices.size(0);
  int thd_sz = tensor.size(1);
  const auto indices_a =
      indices.packed_accessor<int64_t, 1, torch::RestrictPtrTraits, size_t>();

  AT_DISPATCH_ALL_TYPES(
      tensor.scalar_type(), "CUDAIndexPut", ([&] {
        KernIndexPut<scalar_t><<<blk_sz, thd_sz>>>(
            indices_a,
            tensor.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits,
                                   size_t>(),
            reinterpret_cast<scalar_t *>(buffer_data));
      }));
  CudaCheck();
  CudaSafeCall(hipDeviceSynchronize());
}

void CUDAIndexSelect(const torch::Tensor &indices, const void *gl_data,
                     torch::Tensor &tensor) {
  const int blk_sz = indices.size(0);
  const int thd_sz = tensor.size(1);

  const auto indices_a =
      indices.packed_accessor<int64_t, 1, torch::RestrictPtrTraits, size_t>();
  AT_DISPATCH_ALL_TYPES(
      tensor.scalar_type(), "CUDAIndexSelect", ([&] {
        KernIndexSelect<scalar_t><<<blk_sz, thd_sz>>>(
            indices_a, reinterpret_cast<const scalar_t *>(gl_data),
            tensor.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits,
                                   size_t>());
      }));
  CudaCheck();
  CudaSafeCall(hipDeviceSynchronize());
}
}  // namespace tenviz